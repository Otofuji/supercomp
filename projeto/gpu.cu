#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>
#include <vector>
#include <string>
#include<algorithm>
#include<random>
#include <omp.h>

struct matriz { //Com base em exemplo do professor e em https://people.engr.tamu.edu/djimenez/ut/utsa/cs1723/lecture3.html
    int x;
    int y;
    double max;
};

struct sw
{
    int m, n;
    sw(int m_) : m(m_) {};
    
    __host__ __device__
    int operator()(const char& a, const char& b) {
            char notation;
            char tempa = " ";
            char tempb = " ";
            int maximum = 0;
            int maxi = 0;
            int maxj = 0;
            a = " " + a;
            b = " " + b; 
            n = n + 1;
            m = m + 1;
        //PRECISAMOS ALOCAR DINAMICAMENTE ESSA MATRIZ PARA NAO HAVER ESTOURO DE MEMORIA
            matriz* alinhamento[m];
            for (int i = 0; i < m; i++)
                alinhamento[i] = (matriz*)malloc(n * sizeof(matriz));
            for (int i = 0; i < m; i++){
                for (int j = 0; j < n; j++){
                    alinhamento[i][j].max = 0; //Por conveniência, a matriz inteira já foi inicializada com zeros       
                    alinhamento[i][j].x = 0;
                    alinhamento[i][j].y = 0; 
                }
            }
            for (int i = 1; i < m; i++){
                for (int j = 1; j < n; j++) {
                // cout << "POSICAO " << i << " " << j << " CHARS " << a[i] << " " << b[j] << " VALOR ";
                    if (a[i] == b[j]) {
                        if (alinhamento[i-1][j-1].max +2 >= alinhamento[i][j-1].max - 1 && alinhamento[i-1][j-1].max +2  >= alinhamento[i-1][j].max - 1) {
                            alinhamento[i][j].x = i-1;
                            alinhamento[i][j].y = j-1;                    
                            alinhamento[i][j].max = alinhamento[i-1][j-1].max + 2;
                        }
                        else if (alinhamento[i][j-1].max - 1 > alinhamento[i-1][j].max - 1) {
                                    alinhamento[i][j].x = i;
                                    alinhamento[i][j].y = j-1;
                                    alinhamento[i][j].max = alinhamento[i][j-1].max - 1;
                        } else {
                                    alinhamento[i][j].x = i-1;
                                    alinhamento[i][j].y = j;    
                                    alinhamento[i][j].max = alinhamento[i-1][j].max - 1;
                        }                                
                    }
                    else {            
                        if (alinhamento[i-1][j-1].max -1 >= alinhamento[i][j-1].max -1 && alinhamento[i-1][j-1].max -1 >= alinhamento[i-1][j].max - 1) {
                            alinhamento[i][j].x = i-1;
                            alinhamento[i][j].y = j-1;
                            alinhamento[i][j].max = alinhamento[i-1][j-1].max -1 ;
                        }
                        else if (alinhamento[i][j-1].max - 1 > alinhamento[i-1][j].max - 1) {
                                    alinhamento[i][j].x = i;
                                    alinhamento[i][j].y = j-1;
                                    alinhamento[i][j].max = alinhamento[i][j-1].max - 1;
                        } else {
                                    alinhamento[i][j].x = i-1;
                                    alinhamento[i][j].y = j;    
                                    alinhamento[i][j].max = alinhamento[i-1][j].max - 1;                
                        }                                
                    }
                    if (alinhamento[i][j].max > maximum) {
                        maximum = alinhamento[i][j].max;
                        maxi = i;
                        maxj = j;
                    }
                    if (alinhamento[i][j].max < 0) {
                            alinhamento[i][j].max = 0;
                        }
                }
            }    
            //Match = +2
            //Mismatch = -1
            //Gap = -1
            return maximum;
    }
};

int main() {
    // Alocação do vetor na CPU e leitura de dados da entrada-padrão
    int n;
    int m;
    std::cin >> m >> n;
    thrust::host_vector<double> a(m);
    for(int i=0;i<m;i++)
       std::cin>>a[i];
    thrust::host_vector<double> b(n);
    for(int i=0;i<n;i++)
       std::cin>>b[i];
    // Alocação do vetor na GPU e inicialização de dados
    thrust::device_vector<double> a(a);
    thrust::device_vector<double> b(b);
    //inicial do algoritmo de busca exaustiva
    char sa;
    char sb;
    int max_for_now = 0;
    int max = 0;
    //loop da busca exaustiva
    //a ideia aqui é aproveitar a lógica da busca exaustiva, mas implementando em gpu para paralelizar usando thrust

    return max;
}