#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>
#include <vector>
#include <string>
#include<algorithm>
#include<random>
#include <omp.h>

int smith_waterman(int m, int n, string a, string b) {
    //REFERÊNCIAS
    //Martin Mann, Mostafa M Mohamed, Syed M Ali, and Rolf Backofen
    //Interactive implementations of thermodynamics-based RNA structure and RNA-RNA interaction prediction approaches for example-driven teaching
    //PLOS Computational Biology, 14 (8), e1006341, 2018.
    //Martin Raden, Syed M Ali, Omer S Alkhnbashi, Anke Busch, Fabrizio Costa, Jason A Davis, Florian Eggenhofer, Rick Gelhausen, Jens Georg, Steffen Heyne, Michael Hiller, Kousik Kundu, Robert Kleinkauf, Steffen C Lott, Mostafa M Mohamed, Alexander Mattheis, Milad Miladi, Andreas S Richter, Sebastian Will, Joachim Wolff, Patrick R Wright, and Rolf Backofen
    //Freiburg RNA tools: a central online resource for RNA-focused research and teaching
    //Nucleic Acids Research, 46(W1), W25-W29, 2018.

    //Usar matriz clássica do C++ cm struct
    //Criando uma matriz clássica do C++ usando struct, tendo por base exemplo do professor e https://people.engr.tamu.edu/djimenez/ut/utsa/cs1723/lecture3.html
   
    string notation;
    string tempa = " ";
    string tempb = " ";
    int maximum = 0;
    int maxi = 0;
    int maxj = 0;
 
    
    a = " " + a;
    b = " " + b; 

    n = n + 1;

    m = m + 1;
   // matriz alinhamento[m][n]; //ARRUMEI A POSICAO
   
   //PRECISAMOS ALOCAR DINAMICAMENTE ESSA MATRIZ PARA NAO HAVER ESTOURO DE MEMORIA
    matriz* alinhamento[m];
    for (int i = 0; i < m; i++)
        alinhamento[i] = (matriz*)malloc(n * sizeof(matriz));
 
    


    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            alinhamento[i][j].max = 0; //Por conveniência, a matriz inteira já foi inicializada com zeros       
           
            alinhamento[i][j].x = 0;
            alinhamento[i][j].y = 0; 
        }
    }

    for (int i = 1; i < m; i++){
        for (int j = 1; j < n; j++) {
           // cout << "POSICAO " << i << " " << j << " CHARS " << a[i] << " " << b[j] << " VALOR ";
            if (a[i] == b[j]) {
                           

                if (alinhamento[i-1][j-1].max +2 >= alinhamento[i][j-1].max - 1 && alinhamento[i-1][j-1].max +2  >= alinhamento[i-1][j].max - 1) {
                
                    alinhamento[i][j].x = i-1;
                    alinhamento[i][j].y = j-1;
                   
                    alinhamento[i][j].max = alinhamento[i-1][j-1].max + 2;
                    
                }
                else if (alinhamento[i][j-1].max - 1 > alinhamento[i-1][j].max - 1) {
                       
                            alinhamento[i][j].x = i;
                            alinhamento[i][j].y = j-1;
                            alinhamento[i][j].max = alinhamento[i][j-1].max - 1;
                        
                } else {
                        
                            alinhamento[i][j].x = i-1;
                            alinhamento[i][j].y = j;    
                            alinhamento[i][j].max = alinhamento[i-1][j].max - 1;
                                           
                }                                
                
                
            } 

            else {            
                if (alinhamento[i-1][j-1].max -1 >= alinhamento[i][j-1].max -1 && alinhamento[i-1][j-1].max -1 >= alinhamento[i-1][j].max - 1) {
                
                    alinhamento[i][j].x = i-1;
                    alinhamento[i][j].y = j-1;

                   
                    alinhamento[i][j].max = alinhamento[i-1][j-1].max -1 ;
                    
                }
                else if (alinhamento[i][j-1].max - 1 > alinhamento[i-1][j].max - 1) {
                       
                            alinhamento[i][j].x = i;
                            alinhamento[i][j].y = j-1;
                            alinhamento[i][j].max = alinhamento[i][j-1].max - 1;
                        
                } else {
                        
                            alinhamento[i][j].x = i-1;
                            alinhamento[i][j].y = j;    
                            alinhamento[i][j].max = alinhamento[i-1][j].max - 1;
                                           
                }                                
                
            }
                
            if (alinhamento[i][j].max > maximum) {
                maximum = alinhamento[i][j].max;
                maxi = i;
                maxj = j;
            }
            if (alinhamento[i][j].max < 0) {
                    alinhamento[i][j].max = 0;
                }

         //   cout << alinhamento[i][j].max << " APONTANDO PARA " << alinhamento[i][j].x << " " << alinhamento[i][j].y << endl;
            
        }
    }    
    
    
    //Match = +2
    //Mismatch = -1
    //Gap = -1

    return maximum;
}


int main() {
    // Alocação do vetor na CPU e leitura de dados da entrada-padrão
    int n;
    int m;
    std::cin >> m >> n;
    thrust::host_vector<double> a(m);
    for(int i=0;i<m;i++)
       std::cin>>a[i];

    thrust::host_vector<double> b(n);
    for(int i=0;i<n;i++)
       std::cin>>b[i];

    // Alocação do vetor na GPU e inicialização de dados
    thrust::device_vector<double> a(a);
    thrust::device_vector<double> b(b); 
    string sa;
    string sb;
    int max_for_now = 0;
    int max = 0;

    #pragma omp parallel for
    for (int i = 0; i < m; i++) {
        #pragma omp parallel for
        for (int j = 0; j < n; j++) {
            #pragma omp parallel for
            for (int k = 0; k < m || k < n; k++) {
                if (k < a.size()) sa = a.substr(k, m);
                if (k < b.size()) sb = b.substr(k, n);
                max_for_now = smith_waterman(sa.size(), sb.size(), sa, sb);
                while (max_for_now > max) {
                    max++;
                }
            }
        }
    }

    
    // Percurso do vetor na GPU - LENTO
    std::cout <<"\nGPU: ";
    for (thrust::device_vector<double>::iterator elem = vgpu.begin(); elem != vgpu.end(); elem++) {
        std::cout << *elem << " ";
    } 

    // Soma todos os elementos do vetor na GPU
    double soma=thrust::reduce(vgpu.begin(), vgpu.end(), (double) 0, thrust::plus<double>());
    std::cout<<"\nSoma: "<< soma;

    return max;
}
