#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/random.h>
#include <thrust/iterator/counting_iterator.h>
#include <iostream>
#include <vector>
#include <string>
#include<algorithm>
#include <omp.h>


int main() {
    // Alocação do vetor na CPU e leitura de dados da entrada-padrão
    int n;
    int m;
    std::cin >> m >> n;
    thrust::host_vector<double> a(m);
    for(int i=0;i<m;i++)
       std::cin>>a[i];
    thrust::host_vector<double> b(n);
    for(int i=0;i<n;i++)
       std::cin>>b[i];
    // Alocação do vetor na GPU e inicialização de dados
    thrust::device_vector<double> aa(a);
    thrust::device_vector<double> bb(b);

    ////1. Gerar uma subsequencia sb=b[j,j+1,...,j+k] de b, de tamanho aleatório k, 1<=k<=m, e 0<=j<=m 
    thrust::default_random_engine generator1(1);
    thrust::uniform_real_distribution<double> distribution1(1, m);
    int k = distribution1(generator1);
    while (k < m*2) k = k*2;
    k = k/3;
    thrust::default_random_engine generator2(1);
    thrust::uniform_real_distribution<double> distribution2(0, m);
    int j = distribution2(generator2);
    while (j < m*2) j = j*2;
    j = j/3;
    
    thrust::default_random_engine generator3(1);
    thrust::uniform_real_distribution<double> distribution3(m+n, m*n);
    int p = distribution3(generator3);
    int max = 0;
    int max_for_now;
    for (int q = 0; q < p; q++) {
        int a_size;
        if (q+k < m-1) {
            a_size = q+k;
        }
        else {
            if (q < m-1) {
                a_size = m-2;
            }
            else break;
        }

        /*
---------------------------------------------------------------------
        */
        
        // thrust::counting_iterator<int>c0(1);
        // thrust::counting_iterator<int>c1(m+1);
        // thrust::device vector<int> calc[2];
        // calc[0].resize(N+1);
        // calc[1].resize(N+1);
        // thrust::fill(calc[0].begin(),calc[0].end(),0);
        // thrust::device_vector<char> d_s(n);
        // for (int i = 0; i < n; i++) {
        //     d_s[i] = s[i];
        // };
        // for (int j = 0; j < m; j++) {
        //     char letradeT = T[j];
        //     thrust::transform(c0, c1, calc[1].begin() + 1, meu_functor(d_s.data(), letradeT, calc[0].data()));
        //     thrust::inclusive_scan(calc[1].begin()+1, calc[1].end(), calc[0].begin()+1, thurst::maximum<int>());
        // };

        // thrust::reduce(calc[0].begin()+1, calc[0].end(), 1, thrust::maximum<int>());

        /*
---------------------------------------------------------------------
        */

        while (max_for_now > max) {
            max++;
        }

        std::cout << max;
    }

    return 0;
}