#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/random.h>
#include <thrust/iterator/counting_iterator.h>
#include <iostream>
#include <vector>
#include <string>
#include<algorithm>
#include <omp.h>

int main() {
    // Alocação do vetor na CPU e leitura de dados da entrada-padrão
    int n;
    int m;
    std::cin >> m >> n;
    thrust::host_vector<double> a(m);
    for(int i=0;i<m;i++)
       std::cin>>a[i];
    thrust::host_vector<double> b(n);
    for(int i=0;i<n;i++)
       std::cin>>b[i];
    // Alocação do vetor na GPU e inicialização de dados
    thrust::device_vector<double> aa(a);
    thrust::device_vector<double> bb(b);

    ////1. Gerar uma subsequencia sb=b[j,j+1,...,j+k] de b, de tamanho aleatório k, 1<=k<=m, e 0<=j<=m 
    thrust::default_random_engine generator1(1);
    thrust::uniform_real_distribution<double> distribution1(1, m);
    int k = distribution1(generator1);
    while (k < m*2) k = k*2;
    k = k/3;
    thrust::default_random_engine generator2(1);
    thrust::uniform_real_distribution<double> distribution2(0, m);
    int j = distribution2(generator2);
    while (j < m*2) j = j*2;
    j = j/3;
    
    thrust::default_random_engine generator3(1);
    thrust::uniform_real_distribution<double> distribution3(m+n, m*n);
    int p = distribution3(generator3);
    int max = 0;
    int max_for_now;
    for (int q = 0; q < p; q++) {
        int a_size;
        if (q+k < m-1) {
            a_size = q+k;
        }
        else {
            if (q < m-1) {
                a_size = m-2;
            }
            else break;
        }

        thrust::transform(aa.begin()+q, aa.begin()+a_size, )

        while (max_for_now > max) {
            max++;
        }

        std::cout << max;
    }

    return 0;
}